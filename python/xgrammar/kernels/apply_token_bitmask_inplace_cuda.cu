#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

int32_t constexpr kBitsPerMaskElement = 32;
int32_t constexpr kThreadsPerBlock = 256;

template <typename T>
__device__ T GetNegativeInfinity() {
  return -INFINITY;
}

template <>
__device__ half GetNegativeInfinity<half>() {
  return __float2half(-INFINITY);
}

template <>
__device__ __hip_bfloat16 GetNegativeInfinity<__hip_bfloat16>() {
  return __float2bfloat16(-INFINITY);
}

template <typename T, typename PackedT>
__global__ void __launch_bounds__(kThreadsPerBlock) logitsBitmaskKernel(
    T* __restrict__ logits,
    int32_t const* __restrict__ bitmask,
    int32_t const* __restrict__ indices,
    int32_t vocabSize,
    int32_t bitmaskSize
) {
  int constexpr kAlignment = sizeof(PackedT) / sizeof(T);
  int const batchIdx = (indices == nullptr) ? blockIdx.y : indices[blockIdx.y];

  int const logitsGmemOffset = kThreadsPerBlock * blockIdx.x * kBitsPerMaskElement;
  T* logitsGmemPtr = logits + batchIdx * vocabSize + logitsGmemOffset;
  __shared__ T logitsSmem[kThreadsPerBlock * kBitsPerMaskElement];

#pragma unroll
  for (int offset = 0; offset < kThreadsPerBlock * kBitsPerMaskElement;
       offset += kThreadsPerBlock * kAlignment) {
    int localOffset = offset + threadIdx.x * kAlignment;
    if (logitsGmemOffset + localOffset >= vocabSize) {
      break;
    }
    *reinterpret_cast<PackedT*>(logitsSmem + localOffset) =
        *reinterpret_cast<PackedT*>(logitsGmemPtr + localOffset);
  }
  __syncthreads();

  int const bitmaskIdx = kThreadsPerBlock * blockIdx.x + threadIdx.x;
  int32_t const bitmaskVal = bitmask[batchIdx * bitmaskSize + bitmaskIdx];

#pragma unroll
  for (int i = 0; i < kBitsPerMaskElement; ++i) {
    int offset = (i + threadIdx.x) % warpSize;
    if (bitmaskIdx * kBitsPerMaskElement + offset >= vocabSize) {
      continue;
    }
    if (!((bitmaskVal >> offset) & 1)) {
      logitsSmem[threadIdx.x * kBitsPerMaskElement + offset] = GetNegativeInfinity<T>();
    }
  }
  __syncthreads();

#pragma unroll
  for (int offset = 0; offset < kThreadsPerBlock * kBitsPerMaskElement;
       offset += kThreadsPerBlock * kAlignment) {
    int localOffset = offset + threadIdx.x * kAlignment;
    if (logitsGmemOffset + localOffset >= vocabSize) {
      break;
    }
    *reinterpret_cast<PackedT*>(logitsGmemPtr + localOffset) =
        *reinterpret_cast<PackedT*>(logitsSmem + localOffset);
  }
}

template <typename T, typename = std::enable_if_t<std::is_integral<T>::value>>
auto constexpr ceilDiv(T numerator, T denominator) {
  return (numerator + denominator - 1) / denominator;
}

template <typename T>
void applyTokenBitmaskInplaceDispatchToPackedT(
    T* __restrict__ logits,
    int32_t const* __restrict__ bitmask,
    int32_t const* __restrict__ indices,
    int32_t vocabSize,
    int32_t bitmaskSize,
    int32_t batchSize
) {
  dim3 const grid(ceilDiv(bitmaskSize, kThreadsPerBlock), batchSize);
  dim3 const block(kThreadsPerBlock);

  hipStream_t stream = 0;
  if (vocabSize % (sizeof(float4) / sizeof(T)) == 0) {
    logitsBitmaskKernel<T, float4>
        <<<grid, block, 0, stream>>>(logits, bitmask, indices, vocabSize, bitmaskSize);
  } else if (vocabSize % (sizeof(float2) / sizeof(T)) == 0) {
    logitsBitmaskKernel<T, float2>
        <<<grid, block, 0, stream>>>(logits, bitmask, indices, vocabSize, bitmaskSize);
  } else if (vocabSize % (sizeof(float) / sizeof(T)) == 0) {
    logitsBitmaskKernel<T, float>
        <<<grid, block, 0, stream>>>(logits, bitmask, indices, vocabSize, bitmaskSize);
  } else {
    logitsBitmaskKernel<T, T>
        <<<grid, block, 0, stream>>>(logits, bitmask, indices, vocabSize, bitmaskSize);
  }
}

void applyTokenBitmaskInplace(
    at::Tensor logits, at::Tensor bitmask, at::optional<at::Tensor> indices = at::nullopt
) {
  TORCH_CHECK(logits.is_cuda(), "logits must be a CUDA tensor.");
  TORCH_CHECK(logits.is_contiguous(), "logits must be contiguous.");
  TORCH_CHECK(logits.dim() == 1 || logits.dim() == 2, "logits must be a 1D or 2D tensor.");
  int32_t batchSize = 1;
  int32_t vocabSize = logits.size(0);
  if (logits.dim() == 2) {
    batchSize = logits.size(0);
    vocabSize = logits.size(1);
  }

  TORCH_CHECK(bitmask.is_cuda(), "bitmask must be a CUDA tensor.");
  TORCH_CHECK(bitmask.is_contiguous(), "bitmask must be contiguous.");
  TORCH_CHECK(bitmask.dim() == 1 || bitmask.dim() == 2, "bitmask must be a 1D or 2D tensor.");
  int32_t bitmaskBatchSize = 1;
  int32_t bitmaskSize = bitmask.size(0);
  if (bitmask.dim() == 2) {
    bitmaskBatchSize = bitmask.size(0);
    bitmaskSize = bitmask.size(1);
  }
  TORCH_CHECK(bitmaskBatchSize == batchSize, "bitmask must have the batch size same to logits.");
  TORCH_CHECK(
      bitmaskSize == ceilDiv(vocabSize, kBitsPerMaskElement),
      "bitmask must have the hidden size equal to ceilDiv(vocabSize, 32)."
  );

  int32_t* indices_ptr = nullptr;
  if (indices) {
    batchSize = indices->size(0);
    indices_ptr = indices->data_ptr<int32_t>();
  }

  switch (logits.scalar_type()) {
    case torch::kFloat32: {
      applyTokenBitmaskInplaceDispatchToPackedT(
          logits.data_ptr<float>(),
          bitmask.data_ptr<int32_t>(),
          indices_ptr,
          vocabSize,
          bitmaskSize,
          batchSize
      );
      break;
    }
    case torch::kFloat16: {
      applyTokenBitmaskInplaceDispatchToPackedT(
          reinterpret_cast<half*>(logits.data_ptr<torch::Half>()),
          bitmask.data_ptr<int32_t>(),
          indices_ptr,
          vocabSize,
          bitmaskSize,
          batchSize
      );
      break;
    }
    case torch::kBFloat16: {
      applyTokenBitmaskInplaceDispatchToPackedT(
          reinterpret_cast<__hip_bfloat16*>(logits.data_ptr<torch::BFloat16>()),
          bitmask.data_ptr<int32_t>(),
          indices_ptr,
          vocabSize,
          bitmaskSize,
          batchSize
      );
      break;
    }
    default:
      TORCH_CHECK(false, "logits dtype must be float, half or bfloat16.");
      break;
  }
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("apply_token_bitmask_inplace", &applyTokenBitmaskInplace, "Apply token bitmask inplace.");
}
