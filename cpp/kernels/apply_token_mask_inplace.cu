#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <cuda/std/limits>
#include <sstream>

#include "kernels.h"
#include "support/logging.h"

// #ifndef NDEBUG
#define XGRAMMAR_CUDA_CALL(...)                                                                    \
  do {                                                                                             \
    __VA_ARGS__;                                                                                   \
    hipError_t err = hipGetLastError();                                                          \
    XGRAMMAR_CHECK(err != hipSuccess) << "CUDA Error: " << hipGetErrorString(err) << " (" << err \
                                       << ") " << __FILE__ << ": line " << __LINE__ << std::endl;  \
  } while (0)
/*
      return e;                                                                                \
#else
#define XGRAMMAR_CUDA_CALL(func, ...) \
  {                                   \
    hipError_t e = (func);           \
    if (e != hipSuccess) {           \
      return e;                       \
    }                                 \
  }
#endif
*/

#define XGRAMMAR_DISPATCH_DTYPE(dtype_flag, c_type, ...)                                         \
  do {                                                                                           \
    switch (dtype_flag) {                                                                        \
      case DTypeFlag::DTYPE_FLOAT16: {                                                           \
        using c_type = half;                                                                     \
        __VA_ARGS__;                                                                             \
        break;                                                                                   \
      }                                                                                          \
      case DTypeFlag::DTYPE_FLOAT32: {                                                           \
        using c_type = float;                                                                    \
        __VA_ARGS__;                                                                             \
        break;                                                                                   \
      }                                                                                          \
      case DTypeFlag::DTYPE_FLOAT64: {                                                           \
        using c_type = double;                                                                   \
        __VA_ARGS__;                                                                             \
        break;                                                                                   \
      }                                                                                          \
      default:                                                                                   \
        std::ostringstream oss;                                                                  \
        oss << #__VA_ARGS__ << " failed to dispatch data type " << static_cast<int>(dtype_flag); \
        XGRAMMAR_LOG(FATAL) << oss.str();                                                        \
        break;                                                                                   \
    }                                                                                            \
  } while (0)

namespace xgrammar {

#define BITS_PER_BLOCK 32
#define GET_BIT(data_ptr, bit_idx) \
  ((data_ptr[bit_idx / BITS_PER_BLOCK] >> (bit_idx % BITS_PER_BLOCK)) & 1)

template <typename T>
__global__ void __launch_bounds__(1024) apply_token_bitmask_inplace_kernel(
    int* __restrict__ bitmask, T* __restrict__ logits, int batch_size, int vocab_size
) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  int size = batch_size * vocab_size;
  int bitmask_size = (vocab_size + BITS_PER_BLOCK - 1) / BITS_PER_BLOCK;
  for (int i = gid; i < size; i += stride) {
    int batch_id = i / vocab_size;
    int vocab_id = i % vocab_size;
    const int* bitmask_row = bitmask + batch_id * bitmask_size;
    int bit = GET_BIT(bitmask_row, vocab_id);
    logits[gid] = bit ? logits[gid] : -cuda::std::numeric_limits<T>::infinity();
  }
}

#define STRIDE_SIZE 1048576
#define THREADS_PER_BLOCK 1024

void apply_token_bitmask_inplace(
    int* bitmask, void* logits, DTypeFlag dtype_flag, int batch_size, int vocab_size
) {
  int num_blocks = (batch_size * vocab_size + STRIDE_SIZE - 1) / STRIDE_SIZE;
  int num_threads = THREADS_PER_BLOCK;

  XGRAMMAR_DISPATCH_DTYPE(dtype_flag, c_type, {
    XGRAMMAR_CUDA_CALL({
      apply_token_bitmask_inplace_kernel<<<num_blocks, num_threads>>>(
          bitmask, reinterpret_cast<c_type*>(logits), batch_size, vocab_size
      );
    });
  });
}

}  // namespace xgrammar

// PYBIND11_MODULE(ndarray_backend_cuda, m) {
//   namespace py = pybind11;
//   using namespace cuda;

//   m.attr("__device_name__") = "cuda";
//   m.attr("__tile_size__") = TILE;

//   py::class_<CudaArray>(m, "Array")
//       .def(py::init<size_t>(), py::return_value_policy::take_ownership)
//       .def_readonly("size", &CudaArray::size)
//       .def("ptr", &CudaArray::ptr_as_int);

//   // return numpy array, copying from CPU
//   m.def(
//       "to_numpy",
//       [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides, size_t
//       offset ) {
//         std::vector<size_t> numpy_strides = strides;
//         std::transform(
//             numpy_strides.begin(),
//             numpy_strides.end(),
//             numpy_strides.begin(),
//             [](size_t& c) { return c * ELEM_SIZE; }
//         );

//         // copy memory to host
//         scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
//         if (host_ptr == 0) throw std::bad_alloc();
//         hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE,
//         hipMemcpyDeviceToHost); if (err != hipSuccess) throw
//         std::runtime_error(hipGetErrorString(err));

//         // return numpy array
//         py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
//         return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
//       }
//   );

//   // copy numpy array to GPU
//   m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
//     hipError_t err =
//         hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
//     if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
//   });

//   m.def("fill", Fill);
//   m.def("compact", Compact);
//   m.def("ewise_setitem", EwiseSetitem);
//   m.def("scalar_setitem", ScalarSetitem);
//   m.def("ewise_add", EwiseAdd);
//   m.def("scalar_add", ScalarAdd);

//   m.def("ewise_mul", EwiseMul);
//   m.def("scalar_mul", ScalarMul);
//   m.def("ewise_div", EwiseDiv);
//   m.def("scalar_div", ScalarDiv);
//   m.def("scalar_power", ScalarPower);

//   m.def("ewise_maximum", EwiseMaximum);
//   m.def("scalar_maximum", ScalarMaximum);
//   m.def("ewise_eq", EwiseEq);
//   m.def("scalar_eq", ScalarEq);
//   m.def("ewise_ge", EwiseGe);
//   m.def("scalar_ge", ScalarGe);

//   m.def("ewise_log", EwiseLog);
//   m.def("ewise_exp", EwiseExp);
//   m.def("ewise_tanh", EwiseTanh);

//   m.def("matmul", Matmul);

//   m.def("reduce_max", ReduceMax);
//   m.def("reduce_sum", ReduceSum);
// }
