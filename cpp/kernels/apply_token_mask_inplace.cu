#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <cuda/std/limits>
#include <sstream>

#include "../support/logging.h"
#include "kernels.h"

// #ifndef NDEBUG
#define XGRAMMAR_CUDA_CALL(...)                                                                    \
  do {                                                                                             \
    __VA_ARGS__;                                                                                   \
    hipError_t err = hipGetLastError();                                                          \
    XGRAMMAR_CHECK(err == hipSuccess) << "CUDA Error: " << hipGetErrorString(err) << " (" << err \
                                       << ") " << __FILE__ << ": line " << __LINE__ << std::endl;  \
  } while (0)
/*
      return e;                                                                                \
#else
#define XGRAMMAR_CUDA_CALL(func, ...) \
  {                                   \
    hipError_t e = (func);           \
    if (e != hipSuccess) {           \
      return e;                       \
    }                                 \
  }
#endif
*/

#define XGRAMMAR_DISPATCH_DTYPE(dtype_flag, c_type, ...)                                         \
  do {                                                                                           \
    switch (dtype_flag) {                                                                        \
      case DTypeFlag::DTYPE_FLOAT16: {                                                           \
        using c_type = half;                                                                     \
        __VA_ARGS__;                                                                             \
        break;                                                                                   \
      }                                                                                          \
      case DTypeFlag::DTYPE_FLOAT32: {                                                           \
        using c_type = float;                                                                    \
        __VA_ARGS__;                                                                             \
        break;                                                                                   \
      }                                                                                          \
      case DTypeFlag::DTYPE_FLOAT64: {                                                           \
        using c_type = double;                                                                   \
        __VA_ARGS__;                                                                             \
        break;                                                                                   \
      }                                                                                          \
      default:                                                                                   \
        std::ostringstream oss;                                                                  \
        oss << #__VA_ARGS__ << " failed to dispatch data type " << static_cast<int>(dtype_flag); \
        XGRAMMAR_LOG(FATAL) << oss.str();                                                        \
        break;                                                                                   \
    }                                                                                            \
  } while (0)

namespace xgrammar {

#define BITS_PER_BLOCK 32
#define GET_BIT(data_ptr, bit_idx) \
  ((data_ptr[bit_idx / BITS_PER_BLOCK] >> (bit_idx % BITS_PER_BLOCK)) & 1)

template <typename T>
__global__ void __launch_bounds__(512) apply_token_bitmask_inplace_kernel(
    T* __restrict__ logits, const int32_t* __restrict__ bitmask, int batch_size, int vocab_size
) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int size = batch_size * vocab_size;
  if (gid >= size) {
    return;
  }
  int bitmask_size = (vocab_size + BITS_PER_BLOCK - 1) / BITS_PER_BLOCK;
  int batch_id = gid / vocab_size;
  int vocab_id = gid % vocab_size;
  const int32_t* bitmask_row = bitmask + batch_id * bitmask_size;
  int bit = GET_BIT(bitmask_row, vocab_id);
  if (!bit) {
    logits[gid] = -cuda::std::numeric_limits<T>::infinity();
  }
}

#define THREADS_PER_BLOCK 512

void apply_token_bitmask_inplace(
    void* logits, DTypeFlag dtype_flag, int32_t* bitmask, int batch_size, int vocab_size
) {
  int num_blocks = (batch_size * vocab_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  int num_threads = THREADS_PER_BLOCK;

  XGRAMMAR_DISPATCH_DTYPE(dtype_flag, c_type, {
    XGRAMMAR_CUDA_CALL({
      apply_token_bitmask_inplace_kernel<<<num_blocks, num_threads>>>(
          reinterpret_cast<c_type*>(logits), bitmask, batch_size, vocab_size
      );
    });
  });
}

}  // namespace xgrammar
